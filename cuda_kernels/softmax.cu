
#include <hip/hip_runtime.h>
// https://docs.cupy.dev/en/stable/user_guide/kernel.html

// TODO: Why cant we access float.h?
#ifndef FLT_MAX
#define FLT_MAX 3.4028235e+38f
#endif

extern "C" {
__global__ void softmax_forward(float* output, const float* input, int rows, int cols) {
    extern __shared__ float shared_data[];
    
    int row = blockIdx.x;
    int tid = threadIdx.x;
    
    float thread_max = -FLT_MAX;
    float thread_sum = 0.0f;
    
    // Find max and compute exp sum
    for (int c = tid; c < cols; c += blockDim.x) {
        float val = input[row * cols + c];
        thread_max = fmaxf(thread_max, val);
    }
    
    // Reduce max within block
    shared_data[tid] = thread_max;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] = fmaxf(shared_data[tid], shared_data[tid + stride]);
        }
        __syncthreads();
    }

    float max_val = shared_data[0];
    __syncthreads();  // Ensure all threads have the max value

    // Compute exp sum
    for (int c = tid; c < cols; c += blockDim.x) {
        float exp_val = expf(input[row * cols + c] - max_val);
        thread_sum += exp_val;
        output[row * cols + c] = exp_val;  // Store intermediate result
    }

    // Reduce sum within block
    shared_data[tid] = thread_sum;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    float sum_exp = shared_data[0];

    // Normalize with sum
    for (int c = tid; c < cols; c += blockDim.x) {
        output[row * cols + c] /= sum_exp;
    }
}


__global__ void softmax_backward(float* grad_input, const float* grad_output, const float* softmax_output, int rows, int cols) {
    extern __shared__ float shared_data[];
    
    int row = blockIdx.x;
    int tid = threadIdx.x;
    
    float thread_sum = 0.0f;
    
    // Compute dot product of grad_output and softmax_output
    for (int c = tid; c < cols; c += blockDim.x) {
        int idx = row * cols + c;
        thread_sum += grad_output[idx] * softmax_output[idx];
    }
    
    // Reduce sum within block
    shared_data[tid] = thread_sum;
    __syncthreads();
    
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }
    
    float sum = shared_data[0];
    
    // Compute gradient
    for (int c = tid; c < cols; c += blockDim.x) {
        int idx = row * cols + c;
        grad_input[idx] = softmax_output[idx] * (grad_output[idx] - sum);
    }
}

}
